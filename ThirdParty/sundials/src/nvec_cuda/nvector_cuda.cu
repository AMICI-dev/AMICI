#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------
 * Programmer(s): Slaven Peles, and Cody J. Balos @ LLNL
 * -----------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2019, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * -----------------------------------------------------------------
 * This is the implementation file for a MPI+CUDA implementation
 * of the NVECTOR package.
 * -----------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>
#include <cmath>

#include <nvector/cuda/Vector.hpp>
#include <nvector/cuda/VectorKernels.cuh>
#include <nvector/cuda/VectorArrayKernels.cuh>
#include <sundials/sundials_mpi.h>

#define ZERO   RCONST(0.0)
#define HALF   RCONST(0.5)
#define ONE    RCONST(1.0)
#define ONEPT5 RCONST(1.5)

extern "C" {

using namespace suncudavec;

/*
 * Type definitions
 */

typedef suncudavec::Vector<realtype, sunindextype> vector_type;

/* ----------------------------------------------------------------
 * Returns vector type ID. Used to identify vector implementation
 * from abstract N_Vector interface.
 */
N_Vector_ID N_VGetVectorID_Cuda(N_Vector v)
{
  return SUNDIALS_NVEC_CUDA;
}

N_Vector N_VNewEmpty_Cuda()
{
  N_Vector v;
  N_Vector_Ops ops;

  /* Create vector */
  v = NULL;
  v = (N_Vector) malloc(sizeof *v);
  if (v == NULL) return(NULL);

  /* Create vector operation structure */
  ops = NULL;
  ops = (N_Vector_Ops) malloc(sizeof(struct _generic_N_Vector_Ops));
  if (ops == NULL) { free(v); return(NULL); }

  ops->nvgetvectorid     = N_VGetVectorID_Cuda;
  ops->nvclone           = N_VClone_Cuda;
  ops->nvcloneempty      = N_VCloneEmpty_Cuda;
  ops->nvdestroy         = N_VDestroy_Cuda;
  ops->nvspace           = N_VSpace_Cuda;
  ops->nvgetarraypointer = NULL;
  ops->nvsetarraypointer = NULL;

  /* standard vector operations */
  ops->nvlinearsum    = N_VLinearSum_Cuda;
  ops->nvconst        = N_VConst_Cuda;
  ops->nvprod         = N_VProd_Cuda;
  ops->nvdiv          = N_VDiv_Cuda;
  ops->nvscale        = N_VScale_Cuda;
  ops->nvabs          = N_VAbs_Cuda;
  ops->nvinv          = N_VInv_Cuda;
  ops->nvaddconst     = N_VAddConst_Cuda;
  ops->nvdotprod      = N_VDotProd_Cuda;
  ops->nvmaxnorm      = N_VMaxNorm_Cuda;
  ops->nvwrmsnormmask = N_VWrmsNormMask_Cuda;
  ops->nvwrmsnorm     = N_VWrmsNorm_Cuda;
  ops->nvmin          = N_VMin_Cuda;
  ops->nvwl2norm      = N_VWL2Norm_Cuda;
  ops->nvl1norm       = N_VL1Norm_Cuda;
  ops->nvcompare      = N_VCompare_Cuda;
  ops->nvinvtest      = N_VInvTest_Cuda;
  ops->nvconstrmask   = N_VConstrMask_Cuda;
  ops->nvminquotient  = N_VMinQuotient_Cuda;

  /* fused vector operations (optional, NULL means disabled by default) */
  ops->nvlinearcombination = NULL;
  ops->nvscaleaddmulti     = NULL;
  ops->nvdotprodmulti      = NULL;

  /* vector array operations (optional, NULL means disabled by default) */
  ops->nvlinearsumvectorarray         = NULL;
  ops->nvscalevectorarray             = NULL;
  ops->nvconstvectorarray             = NULL;
  ops->nvwrmsnormvectorarray          = NULL;
  ops->nvwrmsnormmaskvectorarray      = NULL;
  ops->nvscaleaddmultivectorarray     = NULL;
  ops->nvlinearcombinationvectorarray = NULL;

  /* Attach ops and set content to NULL */
  v->content = NULL;
  v->ops     = ops;

  return(v);
}

#if SUNDIALS_MPI_ENABLED
N_Vector N_VNew_Cuda(MPI_Comm comm,
                     sunindextype local_length,
                     sunindextype global_length)
{
  N_Vector v;

  v = NULL;
  v = N_VNewEmpty_Cuda();
  if (v == NULL) return(NULL);

  v->content = new vector_type(comm, local_length, global_length);

  return(v);
}

N_Vector N_VNewManaged_Cuda(MPI_Comm comm,
                            sunindextype local_length,
                            sunindextype global_length)
{
  N_Vector v;

  v = NULL;
  v = N_VNewEmpty_Cuda();
  if (v == NULL) return(NULL);

  /* create suncudavec::Vector with managed memory */
  v->content = new vector_type(comm, local_length, global_length, true);

  return(v);
}

N_Vector N_VMake_Cuda(MPI_Comm comm,
                      sunindextype local_length, sunindextype global_length,
                      realtype *h_vdata, realtype *d_vdata)
{
  N_Vector v;

  if (h_vdata == NULL || d_vdata == NULL) return(NULL);

  v = NULL;
  v = N_VNewEmpty_Cuda();
  if (v == NULL) return(NULL);

  /* create suncudavec::Vector using the user-provided data arrays */
  v->content = new vector_type(comm, local_length, global_length, false, false, h_vdata, d_vdata);

  return(v);
}

N_Vector N_VMakeManaged_Cuda(MPI_Comm comm,
                             sunindextype local_length, sunindextype global_length,
                             realtype *vdata)
{
  N_Vector v;

  if (vdata == NULL) return(NULL);
  
  v = NULL;
  v = N_VNewEmpty_Cuda();
  if (v == NULL) return(NULL);

  /* create suncudavec::Vector with managed memory using the user-provided data arrays */
  v->content = new vector_type(comm, local_length, global_length, true, false, vdata, vdata);

  return(v);
}
#else
N_Vector N_VNew_Cuda(sunindextype length)
{
  N_Vector v;

  v = NULL;
  v = N_VNewEmpty_Cuda();
  if (v == NULL) return(NULL);

  v->content = new vector_type(length);

  return(v);
}

N_Vector N_VNewManaged_Cuda(sunindextype length)
{
  N_Vector v;

  v = NULL;
  v = N_VNewEmpty_Cuda();
  if (v == NULL) return(NULL);

  /* create suncudavec::Vector with managed memory */
  v->content = new vector_type(length, true);

  return(v);
}

N_Vector N_VMake_Cuda(sunindextype length, realtype *h_vdata, realtype *d_vdata)
{
  N_Vector v;

  if (h_vdata == NULL || d_vdata == NULL) return(NULL);

  v = NULL;
  v = N_VNewEmpty_Cuda();
  if (v == NULL) return(NULL);

  /* create suncudavec::Vector using the user-provided data arrays */
  v->content = new vector_type(length, false, false, h_vdata, d_vdata);

  return(v);
}

N_Vector N_VMakeManaged_Cuda(sunindextype length, realtype *vdata)
{
  N_Vector v;

  if (vdata == NULL) return(NULL);

  v = NULL;
  v = N_VNewEmpty_Cuda();
  if (v == NULL) return(NULL);

  /* create suncudavec::Vector with managed memory using the user-provided data arrays */
  v->content = new vector_type(length, true, false, vdata, vdata);

  return(v);
}
#endif

/* -----------------------------------------------------------------
 * Function to return the global length of the vector.
 */
sunindextype N_VGetLength_Cuda(N_Vector v)
{
  vector_type* xd = static_cast<vector_type*>(v->content);
  return (xd->sizeGlobal());
}


#if SUNDIALS_MPI_ENABLED
/* -----------------------------------------------------------------
 * Function to return the local length of the vector.
 */
sunindextype N_VGetLocalLength_Cuda(N_Vector v)
{
  vector_type* xd = static_cast<vector_type*>(v->content);
  return (xd->size());
}

/* -----------------------------------------------------------------
 * Function to return the MPI communicator for the vector.
 */
MPI_Comm N_VGetMPIComm_Cuda(N_Vector v)
{
  vector_type* xd = static_cast<vector_type*>(v->content);
  return (xd->comm());
}
#endif

/* ----------------------------------------------------------------------------
 * Return pointer to the raw host data
 */

realtype *N_VGetHostArrayPointer_Cuda(N_Vector x)
{
  vector_type* xv = static_cast<vector_type*>(x->content);
  return (xv->host());
}

/* ----------------------------------------------------------------------------
 * Return pointer to the raw device data
 */

realtype *N_VGetDeviceArrayPointer_Cuda(N_Vector x)
{
  vector_type* xv = static_cast<vector_type*>(x->content);
  return (xv->device());
}

/* ----------------------------------------------------------------------------
 * Return a flag indicating if the memory for the vector data is managed
 */
booleantype N_VIsManagedMemory_Cuda(N_Vector x)
{
  vector_type* xv = static_cast<vector_type*>(x->content);
  return (xv->isManaged());
}

/*
 * ----------------------------------------------------------------------------
 * Sets the hipStream_t to use for execution of the CUDA kernels.
 */
void N_VSetCudaStream_Cuda(N_Vector x, hipStream_t *stream)
{
  vector_type* xv = static_cast<vector_type*>(x->content);
  xv->partStream().setStream(*stream);
  xv->partReduce().setStream(*stream);
}

/* ----------------------------------------------------------------------------
 * Copy vector data to the device
 */

void N_VCopyToDevice_Cuda(N_Vector x)
{
  vector_type* xv = static_cast<vector_type*>(x->content);
  xv->copyToDev();
}

/* ----------------------------------------------------------------------------
 * Copy vector data from the device to the host
 */

void N_VCopyFromDevice_Cuda(N_Vector x)
{
  vector_type* xv = static_cast<vector_type*>(x->content);
  xv->copyFromDev();
}

/* ----------------------------------------------------------------------------
 * Function to print the a CUDA-based vector to stdout
 */

void N_VPrint_Cuda(N_Vector x)
{
  N_VPrintFile_Cuda(x, stdout);
}

/* ----------------------------------------------------------------------------
 * Function to print the a CUDA-based vector to outfile
 */

void N_VPrintFile_Cuda(N_Vector x, FILE *outfile)
{
  sunindextype i;
  vector_type* xd = static_cast<vector_type*>(x->content);

  for (i = 0; i < xd->size(); i++) {
#if defined(SUNDIALS_EXTENDED_PRECISION)
    fprintf(outfile, "%35.32Lg\n", xd->host()[i]);
#elif defined(SUNDIALS_DOUBLE_PRECISION)
    fprintf(outfile, "%19.16g\n", xd->host()[i]);
#else
    fprintf(outfile, "%11.8g\n", xd->host()[i]);
#endif
  }
  fprintf(outfile, "\n");

  return;
}


/*
 * -----------------------------------------------------------------
 * implementation of vector operations
 * -----------------------------------------------------------------
 */

N_Vector N_VCloneEmpty_Cuda(N_Vector w)
{
  N_Vector v;
  N_Vector_Ops ops;

  if (w == NULL) return(NULL);

  /* Create vector */
  v = NULL;
  v = (N_Vector) malloc(sizeof *v);
  if (v == NULL) return(NULL);

  /* Create vector operation structure */
  ops = NULL;
  ops = (N_Vector_Ops) malloc(sizeof(struct _generic_N_Vector_Ops));
  if (ops == NULL) { free(v); return(NULL); }

  ops->nvgetvectorid     = w->ops->nvgetvectorid;
  ops->nvclone           = w->ops->nvclone;
  ops->nvcloneempty      = w->ops->nvcloneempty;
  ops->nvdestroy         = w->ops->nvdestroy;
  ops->nvspace           = w->ops->nvspace;
  ops->nvgetarraypointer = w->ops->nvgetarraypointer;
  ops->nvsetarraypointer = w->ops->nvsetarraypointer;

  /* standard vector operations */
  ops->nvlinearsum    = w->ops->nvlinearsum;
  ops->nvconst        = w->ops->nvconst;
  ops->nvprod         = w->ops->nvprod;
  ops->nvdiv          = w->ops->nvdiv;
  ops->nvscale        = w->ops->nvscale;
  ops->nvabs          = w->ops->nvabs;
  ops->nvinv          = w->ops->nvinv;
  ops->nvaddconst     = w->ops->nvaddconst;
  ops->nvdotprod      = w->ops->nvdotprod;
  ops->nvmaxnorm      = w->ops->nvmaxnorm;
  ops->nvwrmsnormmask = w->ops->nvwrmsnormmask;
  ops->nvwrmsnorm     = w->ops->nvwrmsnorm;
  ops->nvmin          = w->ops->nvmin;
  ops->nvwl2norm      = w->ops->nvwl2norm;
  ops->nvl1norm       = w->ops->nvl1norm;
  ops->nvcompare      = w->ops->nvcompare;
  ops->nvinvtest      = w->ops->nvinvtest;
  ops->nvconstrmask   = w->ops->nvconstrmask;
  ops->nvminquotient  = w->ops->nvminquotient;

  /* fused vector operations */
  ops->nvlinearcombination = w->ops->nvlinearcombination;
  ops->nvscaleaddmulti     = w->ops->nvscaleaddmulti;
  ops->nvdotprodmulti      = w->ops->nvdotprodmulti;

  /* vector array operations */
  ops->nvlinearsumvectorarray         = w->ops->nvlinearsumvectorarray;
  ops->nvscalevectorarray             = w->ops->nvscalevectorarray;
  ops->nvconstvectorarray             = w->ops->nvconstvectorarray;
  ops->nvwrmsnormvectorarray          = w->ops->nvwrmsnormvectorarray;
  ops->nvwrmsnormmaskvectorarray      = w->ops->nvwrmsnormmaskvectorarray;
  ops->nvscaleaddmultivectorarray     = w->ops->nvscaleaddmultivectorarray;
  ops->nvlinearcombinationvectorarray = w->ops->nvlinearcombinationvectorarray;

  /* Create content */
  v->content = NULL;
  v->ops  = ops;

  return(v);
}

N_Vector N_VClone_Cuda(N_Vector w)
{
  N_Vector v;
  vector_type* wdat = static_cast<vector_type*>(w->content);
  vector_type* vdat = new vector_type(*wdat);
  v = NULL;
  v = N_VCloneEmpty_Cuda(w);
  if (v == NULL) return(NULL);

  v->content = vdat;

  return(v);
}


void N_VDestroy_Cuda(N_Vector v)
{
  vector_type* x = static_cast<vector_type*>(v->content);
  if (x != NULL) {
    delete x;
    v->content = NULL;
  }

  free(v->ops); v->ops = NULL;
  free(v); v = NULL;

  return;
}

void N_VSpace_Cuda(N_Vector X, sunindextype *lrw, sunindextype *liw)
{
  int npes;
  vector_type* x = static_cast<vector_type*>(X->content);

  SUNMPI_Comm_size(x->comm(), &npes);

  *lrw = x->sizeGlobal();
  *liw = 2*npes;
}

void N_VConst_Cuda(realtype a, N_Vector X)
{
  vector_type *xvec = static_cast<vector_type*>(X->content);
  setConst(a, *xvec);
}

void N_VLinearSum_Cuda(realtype a, N_Vector X, realtype b, N_Vector Y, N_Vector Z)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  const vector_type *yvec = static_cast<vector_type*>(Y->content);
  vector_type *zvec = static_cast<vector_type*>(Z->content);
  linearSum(a, *xvec, b, *yvec, *zvec);
}

void N_VProd_Cuda(N_Vector X, N_Vector Y, N_Vector Z)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  const vector_type *yvec = static_cast<vector_type*>(Y->content);
  vector_type *zvec = static_cast<vector_type*>(Z->content);
  prod(*xvec, *yvec, *zvec);
}

void N_VDiv_Cuda(N_Vector X, N_Vector Y, N_Vector Z)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  const vector_type *yvec = static_cast<vector_type*>(Y->content);
  vector_type *zvec = static_cast<vector_type*>(Z->content);
  div(*xvec, *yvec, *zvec);
}

void N_VScale_Cuda(realtype a, N_Vector X, N_Vector Z)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  vector_type *zvec = static_cast<vector_type*>(Z->content);
  scale(a, *xvec, *zvec);
}

void N_VAbs_Cuda(N_Vector X, N_Vector Z)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  vector_type *zvec = static_cast<vector_type*>(Z->content);
  absVal(*xvec, *zvec);
}

void N_VInv_Cuda(N_Vector X, N_Vector Z)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  vector_type *zvec = static_cast<vector_type*>(Z->content);
  inv(*xvec, *zvec);
}

void N_VAddConst_Cuda(N_Vector X, realtype b, N_Vector Z)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  vector_type *zvec = static_cast<vector_type*>(Z->content);
  addConst(b, *xvec, *zvec);
}

realtype N_VDotProd_Cuda(N_Vector X, N_Vector Y)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  const vector_type *yvec = static_cast<vector_type*>(Y->content);
  SUNMPI_Comm comm = xvec->comm();

  realtype sum = dotProd(*xvec, *yvec);

  realtype gsum = SUNMPI_Allreduce_scalar(sum, 1, comm);
  return gsum;
}

realtype N_VMaxNorm_Cuda(N_Vector X)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  SUNMPI_Comm comm = xvec->comm();

  realtype locmax = maxNorm(*xvec);

  realtype globmax = SUNMPI_Allreduce_scalar(locmax, 2, comm);
  return globmax;
}

realtype N_VWrmsNorm_Cuda(N_Vector X, N_Vector W)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  const vector_type *wvec = static_cast<vector_type*>(W->content);
  const sunindextype Nglob = xvec->sizeGlobal();
  SUNMPI_Comm comm = xvec->comm();

  realtype sum = wL2NormSquare(*xvec, *wvec);

  realtype gsum = SUNMPI_Allreduce_scalar(sum, 1, comm);
  return std::sqrt(gsum/Nglob);
}

realtype N_VWrmsNormMask_Cuda(N_Vector X, N_Vector W, N_Vector Id)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  const vector_type *wvec = static_cast<vector_type*>(W->content);
  const vector_type *ivec = static_cast<vector_type*>(Id->content);
  const sunindextype Nglob = xvec->sizeGlobal();
  SUNMPI_Comm comm = xvec->comm();

  realtype sum = wL2NormSquareMask(*xvec, *wvec, *ivec);

  realtype gsum = SUNMPI_Allreduce_scalar(sum, 1, comm);
  return std::sqrt(gsum/Nglob);
}

realtype N_VMin_Cuda(N_Vector X)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  SUNMPI_Comm comm = xvec->comm();

  realtype locmin = findMin(*xvec);

  realtype globmin = SUNMPI_Allreduce_scalar(locmin, 3, comm);
  return globmin;
}

realtype N_VWL2Norm_Cuda(N_Vector X, N_Vector W)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  const vector_type *wvec = static_cast<vector_type*>(W->content);
  SUNMPI_Comm comm = xvec->comm();

  realtype sum = wL2NormSquare(*xvec, *wvec);

  realtype gsum = SUNMPI_Allreduce_scalar(sum, 1, comm);
  return std::sqrt(gsum);
}

realtype N_VL1Norm_Cuda(N_Vector X)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  SUNMPI_Comm comm = xvec->comm();

  realtype sum = L1Norm(*xvec);

  realtype gsum = SUNMPI_Allreduce_scalar(sum, 1, comm);
  return gsum;
}

void N_VCompare_Cuda(realtype c, N_Vector X, N_Vector Z)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  vector_type *zvec = static_cast<vector_type*>(Z->content);
  compare(c, *xvec, *zvec);
}

booleantype N_VInvTest_Cuda(N_Vector X, N_Vector Z)
{
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  vector_type *zvec = static_cast<vector_type*>(Z->content);
  SUNMPI_Comm comm = xvec->comm();
  
  realtype locmin = invTest(*xvec, *zvec);

  realtype globmin = SUNMPI_Allreduce_scalar(locmin, 3, comm);
  return (globmin < HALF);
}

/*
 * Creates mask for variables violating constraints
 */
booleantype N_VConstrMask_Cuda(N_Vector C, N_Vector X, N_Vector M)
{
  const vector_type *cvec = static_cast<vector_type*>(C->content);
  const vector_type *xvec = static_cast<vector_type*>(X->content);
  vector_type *mvec = static_cast<vector_type*>(M->content);
  SUNMPI_Comm comm = xvec->comm();

  realtype locsum = constrMask(*cvec, *xvec, *mvec);

  realtype globsum = SUNMPI_Allreduce_scalar(locsum, 1, comm);
  return (globsum < HALF);
}

realtype N_VMinQuotient_Cuda(N_Vector num, N_Vector denom)
{
  const vector_type *numvec = static_cast<vector_type*>(num->content);
  const vector_type *denvec = static_cast<vector_type*>(denom->content);
  SUNMPI_Comm comm = numvec->comm();

  realtype locmin = minQuotient(*numvec, *denvec);

  realtype globmin = SUNMPI_Allreduce_scalar(locmin, 3, comm);
  return globmin;
}

/*
 * -----------------------------------------------------------------
 * fused vector operations
 * -----------------------------------------------------------------
 */

int N_VLinearCombination_Cuda(int nvec, realtype* c, N_Vector* X, N_Vector Z)
{
  hipError_t err;
  vector_type** Xv;
  vector_type*  Zv;

  Zv = static_cast<vector_type*>(Z->content);

  Xv = new vector_type*[nvec];
  for (int i=0; i<nvec; i++)
    Xv[i] = static_cast<vector_type*>(X[i]->content);

  err = linearCombination(nvec, c, Xv, Zv);

  delete[] Xv;

  return err == hipSuccess ? 0 : -1;
}

int N_VScaleAddMulti_Cuda(int nvec, realtype* c, N_Vector X, N_Vector* Y,
                           N_Vector* Z)
{
  hipError_t err;
  vector_type*  Xv;
  vector_type** Yv;
  vector_type** Zv;

  Xv = static_cast<vector_type*>(X->content);

  Yv = new vector_type*[nvec];
  for (int i=0; i<nvec; i++)
    Yv[i] = static_cast<vector_type*>(Y[i]->content);

  Zv = new vector_type*[nvec];
  for (int i=0; i<nvec; i++)
    Zv[i] = static_cast<vector_type*>(Z[i]->content);

  err = scaleAddMulti(nvec, c, Xv, Yv, Zv);

  delete[] Yv;
  delete[] Zv;

  return err == hipSuccess ? 0 : -1;
}


int N_VDotProdMulti_Cuda(int nvec, N_Vector x, N_Vector* Y, realtype* dotprods)
{
  hipError_t err;
  SUNMPI_Comm comm;
  vector_type*  Xv;
  vector_type** Yv;

  Xv = static_cast<vector_type*>(x->content);
  comm = Xv->comm();

  Yv = new vector_type*[nvec];
  for (int i=0; i<nvec; i++)
    Yv[i] = static_cast<vector_type*>(Y[i]->content);

  err = dotProdMulti(nvec, Xv, Yv, dotprods);

  delete[] Yv;

  SUNMPI_Allreduce(dotprods, nvec, 1, comm);

  return err == hipSuccess ? 0 : -1;
}



/*
 * -----------------------------------------------------------------------------
 * vector array operations
 * -----------------------------------------------------------------------------
 */

int N_VLinearSumVectorArray_Cuda(int nvec, realtype a, N_Vector* X, realtype b,
                                 N_Vector* Y, N_Vector* Z)
{
  hipError_t err;
  vector_type** Xv;
  vector_type** Yv;
  vector_type** Zv;

  Xv = new vector_type*[nvec];
  for (int i=0; i<nvec; i++)
    Xv[i] = static_cast<vector_type*>(X[i]->content);

  Yv = new vector_type*[nvec];
  for (int i=0; i<nvec; i++)
    Yv[i] = static_cast<vector_type*>(Y[i]->content);

  Zv = new vector_type*[nvec];
  for (int i=0; i<nvec; i++)
    Zv[i] = static_cast<vector_type*>(Z[i]->content);

  err = linearSumVectorArray(nvec, a, Xv, b, Yv, Zv);

  delete[] Xv;
  delete[] Yv;
  delete[] Zv;

  return err == hipSuccess ? 0 : -1;
}


int N_VScaleVectorArray_Cuda(int nvec, realtype* c, N_Vector* X, N_Vector* Z)
{
  hipError_t err;
  vector_type** Xv;
  vector_type** Zv;

  Xv = new vector_type*[nvec];
  for (int i=0; i<nvec; i++)
    Xv[i] = static_cast<vector_type*>(X[i]->content);

  Zv = new vector_type*[nvec];
  for (int i=0; i<nvec; i++)
    Zv[i] = static_cast<vector_type*>(Z[i]->content);

  err = scaleVectorArray(nvec, c, Xv, Zv);

  delete[] Xv;
  delete[] Zv;

  return err == hipSuccess ? 0 : -1;
}


int N_VConstVectorArray_Cuda(int nvec, realtype c, N_Vector* Z)
{
  hipError_t err;
  vector_type** Zv;

  Zv = new vector_type*[nvec];
  for (int i=0; i<nvec; i++)
    Zv[i] = static_cast<vector_type*>(Z[i]->content);

  err = constVectorArray(nvec, c, Zv);

  delete[] Zv;

  return err == hipSuccess ? 0 : -1;
}


int N_VWrmsNormVectorArray_Cuda(int nvec, N_Vector* X, N_Vector* W,
                                realtype* norms)
{
  hipError_t err;
  const vector_type* xvec = static_cast<vector_type*>(X[0]->content);
  vector_type** Xv;
  vector_type** Wv;
  
  SUNMPI_Comm comm = xvec->comm();
  sunindextype N = xvec->sizeGlobal();

  Xv = new vector_type*[nvec];
  for (int k=0; k<nvec; k++)
    Xv[k] = static_cast<vector_type*>(X[k]->content);

  Wv = new vector_type*[nvec];
  for (int k=0; k<nvec; k++)
    Wv[k] = static_cast<vector_type*>(W[k]->content);

  err = wL2NormSquareVectorArray(nvec, Xv, Wv, norms);

  delete[] Xv;
  delete[] Wv;

  SUNMPI_Allreduce(norms, nvec, 1, comm);

  for (int k=0; k<nvec; ++k) {
    norms[k] = std::sqrt(norms[k]/N);
  }

  return err == hipSuccess ? 0 : -1;
}


int N_VWrmsNormMaskVectorArray_Cuda(int nvec, N_Vector* X, N_Vector* W,
                                    N_Vector id, realtype* norms)
{
  hipError_t err;
  const vector_type* xvec = static_cast<vector_type*>(X[0]->content);
  vector_type** Xv;
  vector_type** Wv;
  vector_type*  IDv;
  
  SUNMPI_Comm comm = xvec->comm();
  sunindextype N = xvec->sizeGlobal();

  Xv = new vector_type*[nvec];
  for (int k=0; k<nvec; k++)
    Xv[k] = static_cast<vector_type*>(X[k]->content);

  Wv = new vector_type*[nvec];
  for (int k=0; k<nvec; k++)
    Wv[k] = static_cast<vector_type*>(W[k]->content);

  IDv = static_cast<vector_type*>(id->content);

  err = wL2NormSquareMaskVectorArray(nvec, Xv, Wv, IDv, norms);

  delete[] Xv;
  delete[] Wv;

  SUNMPI_Allreduce(norms, nvec, 1, comm);

  for (int k=0; k<nvec; ++k) {
    norms[k] = std::sqrt(norms[k]/N);
  }

  return err == hipSuccess ? 0 : -1;
}


int N_VScaleAddMultiVectorArray_Cuda(int nvec, int nsum, realtype* c,
                                      N_Vector* X, N_Vector** Y, N_Vector** Z)
{
  hipError_t err;
  vector_type** Xv;
  vector_type** Yv;
  vector_type** Zv;

  Xv = new vector_type*[nvec];
  for (int k=0; k<nvec; k++)
    Xv[k] = static_cast<vector_type*>(X[k]->content);

  Yv = new vector_type*[nsum*nvec];
  for (int k=0; k<nvec; k++)
    for (int j=0; j<nsum; j++)
      Yv[k*nsum+j] = static_cast<vector_type*>(Y[j][k]->content);

  Zv = new vector_type*[nsum*nvec];
  for (int k=0; k<nvec; k++)
    for (int j=0; j<nsum; j++)
      Zv[k*nsum+j] = static_cast<vector_type*>(Z[j][k]->content);

  err = scaleAddMultiVectorArray(nvec, nsum, c, Xv, Yv, Zv);

  delete[] Xv;
  delete[] Yv;
  delete[] Zv;

  return err == hipSuccess ? 0 : -1;
}


int N_VLinearCombinationVectorArray_Cuda(int nvec, int nsum, realtype* c,
                                         N_Vector** X, N_Vector* Z)
{
  hipError_t err;
  vector_type** Xv;
  vector_type** Zv;

  Xv = new vector_type*[nsum*nvec];
  for (int k=0; k<nvec; k++)
    for (int j=0; j<nsum; j++)
      Xv[k*nsum+j] = static_cast<vector_type*>(X[j][k]->content);

  Zv = new vector_type*[nvec];
  for (int k=0; k<nvec; k++)
    Zv[k] = static_cast<vector_type*>(Z[k]->content);

  err = linearCombinationVectorArray(nvec, nsum, c, Xv, Zv);

  delete[] Xv;
  delete[] Zv;

  return err == hipSuccess ? 0 : -1;
}


/*
 * -----------------------------------------------------------------
 * Enable / Disable fused and vector array operations
 * -----------------------------------------------------------------
 */

int N_VEnableFusedOps_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  if (tf) {
    /* enable all fused vector operations */
    v->ops->nvlinearcombination = N_VLinearCombination_Cuda;
    v->ops->nvscaleaddmulti     = N_VScaleAddMulti_Cuda;
    v->ops->nvdotprodmulti      = N_VDotProdMulti_Cuda;
    /* enable all vector array operations */
    v->ops->nvlinearsumvectorarray         = N_VLinearSumVectorArray_Cuda;
    v->ops->nvscalevectorarray             = N_VScaleVectorArray_Cuda;
    v->ops->nvconstvectorarray             = N_VConstVectorArray_Cuda;
    v->ops->nvwrmsnormvectorarray          = N_VWrmsNormVectorArray_Cuda;
    v->ops->nvwrmsnormmaskvectorarray      = N_VWrmsNormMaskVectorArray_Cuda;
    v->ops->nvscaleaddmultivectorarray     = N_VScaleAddMultiVectorArray_Cuda;
    v->ops->nvlinearcombinationvectorarray = N_VLinearCombinationVectorArray_Cuda;
  } else {
    /* disable all fused vector operations */
    v->ops->nvlinearcombination = NULL;
    v->ops->nvscaleaddmulti     = NULL;
    v->ops->nvdotprodmulti      = NULL;
    /* disable all vector array operations */
    v->ops->nvlinearsumvectorarray         = NULL;
    v->ops->nvscalevectorarray             = NULL;
    v->ops->nvconstvectorarray             = NULL;
    v->ops->nvwrmsnormvectorarray          = NULL;
    v->ops->nvwrmsnormmaskvectorarray      = NULL;
    v->ops->nvscaleaddmultivectorarray     = NULL;
    v->ops->nvlinearcombinationvectorarray = NULL;
  }

  /* return success */
  return(0);
}


int N_VEnableLinearCombination_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvlinearcombination = N_VLinearCombination_Cuda;
  else
    v->ops->nvlinearcombination = NULL;

  /* return success */
  return(0);
}

int N_VEnableScaleAddMulti_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvscaleaddmulti = N_VScaleAddMulti_Cuda;
  else
    v->ops->nvscaleaddmulti = NULL;

  /* return success */
  return(0);
}

int N_VEnableDotProdMulti_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvdotprodmulti = N_VDotProdMulti_Cuda;
  else
    v->ops->nvdotprodmulti = NULL;

  /* return success */
  return(0);
}

int N_VEnableLinearSumVectorArray_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvlinearsumvectorarray = N_VLinearSumVectorArray_Cuda;
  else
    v->ops->nvlinearsumvectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableScaleVectorArray_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvscalevectorarray = N_VScaleVectorArray_Cuda;
  else
    v->ops->nvscalevectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableConstVectorArray_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvconstvectorarray = N_VConstVectorArray_Cuda;
  else
    v->ops->nvconstvectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableWrmsNormVectorArray_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvwrmsnormvectorarray = N_VWrmsNormVectorArray_Cuda;
  else
    v->ops->nvwrmsnormvectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableWrmsNormMaskVectorArray_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvwrmsnormmaskvectorarray = N_VWrmsNormMaskVectorArray_Cuda;
  else
    v->ops->nvwrmsnormmaskvectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableScaleAddMultiVectorArray_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvscaleaddmultivectorarray = N_VScaleAddMultiVectorArray_Cuda;
  else
    v->ops->nvscaleaddmultivectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableLinearCombinationVectorArray_Cuda(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvlinearcombinationvectorarray = N_VLinearCombinationVectorArray_Cuda;
  else
    v->ops->nvlinearcombinationvectorarray = NULL;

  /* return success */
  return(0);
}

} // extern "C"
