#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------
 * Programmer(s): Slaven Peles @ LLNL
 * -----------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2019, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * -----------------------------------------------------------------
 * This is the implementation file for a MPI+RAJA implementation
 * of the NVECTOR package.
 * -----------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>

#include <nvector/raja/Vector.hpp>
#include <sundials/sundials_mpi.h>
#include <RAJA/RAJA.hpp>


#define ZERO   RCONST(0.0)
#define HALF   RCONST(0.5)
#define ONE    RCONST(1.0)
#define ONEPT5 RCONST(1.5)

// RAJA defines
#define CUDA_BLOCK_SIZE 256
#define RAJA_NODE_TYPE RAJA::cuda_exec< CUDA_BLOCK_SIZE >
#define RAJA_REDUCE_TYPE RAJA::cuda_reduce< CUDA_BLOCK_SIZE >
#define RAJA_LAMBDA [=] __device__

extern "C" {

using namespace sunrajavec;

// Type defines
typedef sunrajavec::Vector<realtype, sunindextype> vector_type;

// Static constants
static constexpr sunindextype zeroIdx = 0;

/*
 * ----------------------------------------------------------------
 * private accessor/helper functions
 * ----------------------------------------------------------------
 */

static inline sunindextype getLocalLength(N_Vector v)
{
  vector_type* vp = static_cast<vector_type*>(v->content);
  return vp->size();
}

static inline SUNMPI_Comm getMPIComm(N_Vector v)
{
  vector_type* vp = static_cast<vector_type*>(v->content);
  return vp->comm();
}

/* ----------------------------------------------------------------
 * Returns vector type ID. Used to identify vector implementation
 * from abstract N_Vector interface.
 */
N_Vector_ID N_VGetVectorID_Raja(N_Vector v)
{
  return SUNDIALS_NVEC_RAJA;
}

N_Vector N_VNewEmpty_Raja()
{
  N_Vector v;
  N_Vector_Ops ops;

  /* Create vector */
  v = NULL;
  v = (N_Vector) malloc(sizeof *v);
  if (v == NULL) return(NULL);

  /* Create vector operation structure */
  ops = NULL;
  ops = (N_Vector_Ops) malloc(sizeof(struct _generic_N_Vector_Ops));
  if (ops == NULL) { free(v); return(NULL); }

  ops->nvgetvectorid     = N_VGetVectorID_Raja;
  ops->nvclone           = N_VClone_Raja;
  ops->nvcloneempty      = N_VCloneEmpty_Raja;
  ops->nvdestroy         = N_VDestroy_Raja;
  ops->nvspace           = N_VSpace_Raja;
  ops->nvgetarraypointer = NULL; //N_VGetArrayPointer_Raja;
  ops->nvsetarraypointer = NULL; //N_VSetArrayPointer_Raja;

  /* standard vector operations */
  ops->nvlinearsum    = N_VLinearSum_Raja;
  ops->nvconst        = N_VConst_Raja;
  ops->nvprod         = N_VProd_Raja;
  ops->nvdiv          = N_VDiv_Raja;
  ops->nvscale        = N_VScale_Raja;
  ops->nvabs          = N_VAbs_Raja;
  ops->nvinv          = N_VInv_Raja;
  ops->nvaddconst     = N_VAddConst_Raja;
  ops->nvdotprod      = N_VDotProd_Raja;
  ops->nvmaxnorm      = N_VMaxNorm_Raja;
  ops->nvwrmsnormmask = N_VWrmsNormMask_Raja;
  ops->nvwrmsnorm     = N_VWrmsNorm_Raja;
  ops->nvmin          = N_VMin_Raja;
  ops->nvwl2norm      = N_VWL2Norm_Raja;
  ops->nvl1norm       = N_VL1Norm_Raja;
  ops->nvcompare      = N_VCompare_Raja;
  ops->nvinvtest      = N_VInvTest_Raja;
  ops->nvconstrmask   = N_VConstrMask_Raja;
  ops->nvminquotient  = N_VMinQuotient_Raja;

  /* fused vector operations (optional, NULL means disabled by default) */
  ops->nvlinearcombination = NULL;
  ops->nvscaleaddmulti     = NULL;
  ops->nvdotprodmulti      = NULL;

  /* vector array operations (optional, NULL means disabled by default) */
  ops->nvlinearsumvectorarray         = NULL;
  ops->nvscalevectorarray             = NULL;
  ops->nvconstvectorarray             = NULL;
  ops->nvwrmsnormvectorarray          = NULL;
  ops->nvwrmsnormmaskvectorarray      = NULL;
  ops->nvscaleaddmultivectorarray     = NULL;
  ops->nvlinearcombinationvectorarray = NULL;

  /* Attach ops and set content to NULL */
  v->content = NULL;
  v->ops     = ops;

  return(v);
}


#if SUNDIALS_MPI_ENABLED
N_Vector N_VNew_Raja(MPI_Comm comm,
                     sunindextype local_length,
                     sunindextype global_length)
{
  N_Vector v;

  v = NULL;
  v = N_VNewEmpty_Raja();
  if (v == NULL) return(NULL);

  v->content = new vector_type(comm, local_length, global_length);

  return(v);
}
#else
N_Vector N_VNew_Raja(sunindextype length)
{
  N_Vector v;

  v = NULL;
  v = N_VNewEmpty_Raja();
  if (v == NULL) return(NULL);

  v->content = new vector_type(length);

  return(v);
}
#endif


N_Vector N_VMake_Raja(N_VectorContent_Raja c)
{
  N_Vector v;
  vector_type* x = static_cast<vector_type*>(c);
  sunindextype length = x->size();

  v = NULL;
  v = N_VNewEmpty_Raja();
  if (v == NULL) return(NULL);

  v->content = c;

  return(v);
}


/* -----------------------------------------------------------------
 * Function to return the global length of the vector.
 */
sunindextype N_VGetLength_Raja(N_Vector v)
{
  vector_type* xd = static_cast<vector_type*>(v->content);
  return xd->sizeGlobal();
}

#if SUNDIALS_MPI_ENABLED
/* -----------------------------------------------------------------
 * Function to return the local length of the vector.
 */
sunindextype N_VGetLocalLength_Raja(N_Vector v)
{
  vector_type* xd = static_cast<vector_type*>(v->content);
  return xd->size();
}

/* -----------------------------------------------------------------
 * Function to return the MPI communicator for the vector.
 */
MPI_Comm N_VGetMPIComm_Raja(N_Vector v)
{
  vector_type* xd = static_cast<vector_type*>(v->content);
  return (xd->comm());
}
#endif

/* ----------------------------------------------------------------------------
 * Return pointer to the raw host data
 */

realtype *N_VGetHostArrayPointer_Raja(N_Vector x)
{
  vector_type* xv = static_cast<vector_type*>(x->content);
  return (xv->host());
}

/* ----------------------------------------------------------------------------
 * Return pointer to the raw device data
 */

realtype *N_VGetDeviceArrayPointer_Raja(N_Vector x)
{
  vector_type* xv = static_cast<vector_type*>(x->content);
  return (xv->device());
}

/* ----------------------------------------------------------------------------
 * Copy vector data to the device
 */

void N_VCopyToDevice_Raja(N_Vector x)
{
  vector_type* xv = static_cast<vector_type*>(x->content);
  xv->copyToDev();
}

/* ----------------------------------------------------------------------------
 * Copy vector data from the device to the host
 */

void N_VCopyFromDevice_Raja(N_Vector x)
{
  vector_type* xv = static_cast<vector_type*>(x->content);
  xv->copyFromDev();
}

/* ----------------------------------------------------------------------------
 * Function to print the a serial vector to stdout
 */

void N_VPrint_Raja(N_Vector X)
{
  N_VPrintFile_Raja(X, stdout);
}

/* ----------------------------------------------------------------------------
 * Function to print the a serial vector to outfile
 */

void N_VPrintFile_Raja(N_Vector X, FILE *outfile)
{
  const realtype *xd = N_VGetDeviceArrayPointer_Raja(X);
  const sunindextype N = getLocalLength(X);
  sunindextype i;

  for (i = 0; i < N; ++i) {
#if defined(SUNDIALS_EXTENDED_PRECISION)
    fprintf(outfile, "%35.32Lg\n", xd[i]);
#elif defined(SUNDIALS_DOUBLE_PRECISION)
    fprintf(outfile, "%19.16g\n", xd[i]);
#else
    fprintf(outfile, "%11.8g\n", xd[i]);
#endif
  }
  fprintf(outfile, "\n");

  return;
}

/*
 * -----------------------------------------------------------------
 * implementation of vector operations
 * -----------------------------------------------------------------
 */

N_Vector N_VCloneEmpty_Raja(N_Vector w)
{
  N_Vector v;
  N_Vector_Ops ops;

  if (w == NULL) return(NULL);

  /* Create vector */
  v = NULL;
  v = (N_Vector) malloc(sizeof *v);
  if (v == NULL) return(NULL);

  /* Create vector operation structure */
  ops = NULL;
  ops = (N_Vector_Ops) malloc(sizeof(struct _generic_N_Vector_Ops));
  if (ops == NULL) { free(v); return(NULL); }

  ops->nvgetvectorid     = w->ops->nvgetvectorid;
  ops->nvclone           = w->ops->nvclone;
  ops->nvcloneempty      = w->ops->nvcloneempty;
  ops->nvdestroy         = w->ops->nvdestroy;
  ops->nvspace           = w->ops->nvspace;
  ops->nvgetarraypointer = w->ops->nvgetarraypointer;
  ops->nvsetarraypointer = w->ops->nvsetarraypointer;

  /* standard vector operations */
  ops->nvlinearsum    = w->ops->nvlinearsum;
  ops->nvconst        = w->ops->nvconst;
  ops->nvprod         = w->ops->nvprod;
  ops->nvdiv          = w->ops->nvdiv;
  ops->nvscale        = w->ops->nvscale;
  ops->nvabs          = w->ops->nvabs;
  ops->nvinv          = w->ops->nvinv;
  ops->nvaddconst     = w->ops->nvaddconst;
  ops->nvdotprod      = w->ops->nvdotprod;
  ops->nvmaxnorm      = w->ops->nvmaxnorm;
  ops->nvwrmsnormmask = w->ops->nvwrmsnormmask;
  ops->nvwrmsnorm     = w->ops->nvwrmsnorm;
  ops->nvmin          = w->ops->nvmin;
  ops->nvwl2norm      = w->ops->nvwl2norm;
  ops->nvl1norm       = w->ops->nvl1norm;
  ops->nvcompare      = w->ops->nvcompare;
  ops->nvinvtest      = w->ops->nvinvtest;
  ops->nvconstrmask   = w->ops->nvconstrmask;
  ops->nvminquotient  = w->ops->nvminquotient;

  /* fused vector operations */
  ops->nvlinearcombination = w->ops->nvlinearcombination;
  ops->nvscaleaddmulti     = w->ops->nvscaleaddmulti;
  ops->nvdotprodmulti      = w->ops->nvdotprodmulti;

  /* vector array operations */
  ops->nvlinearsumvectorarray         = w->ops->nvlinearsumvectorarray;
  ops->nvscalevectorarray             = w->ops->nvscalevectorarray;
  ops->nvconstvectorarray             = w->ops->nvconstvectorarray;
  ops->nvwrmsnormvectorarray          = w->ops->nvwrmsnormvectorarray;
  ops->nvwrmsnormmaskvectorarray      = w->ops->nvwrmsnormmaskvectorarray;
  ops->nvscaleaddmultivectorarray     = w->ops->nvscaleaddmultivectorarray;
  ops->nvlinearcombinationvectorarray = w->ops->nvlinearcombinationvectorarray;

  /* Create content */
  v->content = NULL;
  v->ops  = ops;

  return(v);
}

N_Vector N_VClone_Raja(N_Vector w)
{
  N_Vector v;
  vector_type* wdat = static_cast<vector_type*>(w->content);
  vector_type* vdat = new vector_type(*wdat);
  v = NULL;
  v = N_VCloneEmpty_Raja(w);
  if (v == NULL) return(NULL);

  v->content = vdat;

  return(v);
}


void N_VDestroy_Raja(N_Vector v)
{
  vector_type* x = static_cast<vector_type*>(v->content);
  if (x != NULL) {
    delete x;
    v->content = NULL;
  }

  free(v->ops); v->ops = NULL;
  free(v); v = NULL;

  return;
}

void N_VSpace_Raja(N_Vector X, sunindextype *lrw, sunindextype *liw)
{
  SUNMPI_Comm comm = getMPIComm(X);
  int npes;

  SUNMPI_Comm_size(comm, &npes);

  *lrw = N_VGetLength_Raja(X);
  *liw = 2*npes;
}

void N_VConst_Raja(realtype c, N_Vector Z)
{
  const sunindextype N = getLocalLength(Z);
  realtype *zdata = N_VGetDeviceArrayPointer_Raja(Z);

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N), RAJA_LAMBDA(sunindextype i) {
     zdata[i] = c;
  });
}

void N_VLinearSum_Raja(realtype a, N_Vector X, realtype b, N_Vector Y, N_Vector Z)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const realtype *ydata = N_VGetDeviceArrayPointer_Raja(Y);
  const sunindextype N = getLocalLength(X);
  realtype *zdata = N_VGetDeviceArrayPointer_Raja(Z);

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      zdata[i] = a*xdata[i] + b*ydata[i];
    }
  );
}

void N_VProd_Raja(N_Vector X, N_Vector Y, N_Vector Z)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const realtype *ydata = N_VGetDeviceArrayPointer_Raja(Y);
  const sunindextype N = getLocalLength(X);
  realtype *zdata = N_VGetDeviceArrayPointer_Raja(Z);

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      zdata[i] = xdata[i] * ydata[i];
    }
  );
}

void N_VDiv_Raja(N_Vector X, N_Vector Y, N_Vector Z)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const realtype *ydata = N_VGetDeviceArrayPointer_Raja(Y);
  const sunindextype N = getLocalLength(X);
  realtype *zdata = N_VGetDeviceArrayPointer_Raja(Z);

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      zdata[i] = xdata[i] / ydata[i];
    }
  );
}

void N_VScale_Raja(realtype c, N_Vector X, N_Vector Z)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const sunindextype N = getLocalLength(X);
  realtype *zdata = N_VGetDeviceArrayPointer_Raja(Z);

  RAJA::forall<RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      zdata[i] = c * xdata[i];
    }
  );
}

void N_VAbs_Raja(N_Vector X, N_Vector Z)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const sunindextype N = getLocalLength(X);
  realtype *zdata = N_VGetDeviceArrayPointer_Raja(Z);

  RAJA::forall<RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      zdata[i] = abs(xdata[i]);
    }
  );
}

void N_VInv_Raja(N_Vector X, N_Vector Z)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const sunindextype N = getLocalLength(X);
  realtype *zdata = N_VGetDeviceArrayPointer_Raja(Z);

  RAJA::forall<RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      zdata[i] = ONE / xdata[i];
    }
  );
}

void N_VAddConst_Raja(N_Vector X, realtype b, N_Vector Z)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const sunindextype N = getLocalLength(X);
  realtype *zdata = N_VGetDeviceArrayPointer_Raja(Z);

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      zdata[i] = xdata[i] + b;
    }
  );
}

realtype N_VDotProd_Raja(N_Vector X, N_Vector Y)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const realtype *ydata = N_VGetDeviceArrayPointer_Raja(Y);
  const sunindextype N = getLocalLength(X);

  RAJA::ReduceSum< RAJA_REDUCE_TYPE, realtype> gpu_result(0.0);
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      gpu_result += xdata[i] * ydata[i] ;
    }
  );

  /* Reduce across MPI processes */
  realtype sum = static_cast<realtype>(gpu_result);
  SUNMPI_Comm comm = getMPIComm(X);
  realtype gsum = SUNMPI_Allreduce_scalar(sum, 1, comm);
  return gsum;
}

realtype N_VMaxNorm_Raja(N_Vector X)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const sunindextype N = getLocalLength(X);

  RAJA::ReduceMax< RAJA_REDUCE_TYPE, realtype> gpu_result(0.0);
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      gpu_result.max(abs(xdata[i]));
    }
  );

  /* Reduce across MPI processes */
  realtype maximum = static_cast<realtype>(gpu_result);
  SUNMPI_Comm comm = getMPIComm(X);
  return SUNMPI_Allreduce_scalar(maximum, 2, comm);
}

realtype N_VWrmsNorm_Raja(N_Vector X, N_Vector W)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const realtype *wdata = N_VGetDeviceArrayPointer_Raja(W);
  const sunindextype N = getLocalLength(X);
  const sunindextype Nglobal = N_VGetLength_Raja(X);

  RAJA::ReduceSum< RAJA_REDUCE_TYPE, realtype> gpu_result(0.0);
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      gpu_result += (xdata[i] * wdata[i] * xdata[i] * wdata[i]);
    }
  );

  /* Reduce across MPI processes */
  realtype sum = static_cast<realtype>(gpu_result);
  SUNMPI_Comm comm = getMPIComm(X);
  return std::sqrt(SUNMPI_Allreduce_scalar(sum, 1, comm)/Nglobal);
}

realtype N_VWrmsNormMask_Raja(N_Vector X, N_Vector W, N_Vector ID)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const realtype *wdata = N_VGetDeviceArrayPointer_Raja(W);
  const realtype *iddata = N_VGetDeviceArrayPointer_Raja(ID);
  const sunindextype N = getLocalLength(X);
  const sunindextype Nglobal = N_VGetLength_Raja(X);

  RAJA::ReduceSum< RAJA_REDUCE_TYPE, realtype> gpu_result(0.0);
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      if (iddata[i] > ZERO)
        gpu_result += (xdata[i] * wdata[i] * xdata[i] * wdata[i]);
    }
  );

  /* Reduce across MPI processes */
  realtype sum = static_cast<realtype>(gpu_result);
  SUNMPI_Comm comm = getMPIComm(X);
  return std::sqrt(SUNMPI_Allreduce_scalar(sum, 1, comm)/Nglobal);
}

realtype N_VMin_Raja(N_Vector X)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const sunindextype N = getLocalLength(X);

  RAJA::ReduceMin< RAJA_REDUCE_TYPE, realtype> gpu_result(std::numeric_limits<realtype>::max());
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      gpu_result.min(xdata[i]);
    }
  );

  /* Reduce across MPI processes */
  realtype minumum = static_cast<realtype>(gpu_result);
  SUNMPI_Comm comm = getMPIComm(X);
  return SUNMPI_Allreduce_scalar(minumum, 3, comm);
}

realtype N_VWL2Norm_Raja(N_Vector X, N_Vector W)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const realtype *wdata = N_VGetDeviceArrayPointer_Raja(W);
  const sunindextype N = getLocalLength(X);

  RAJA::ReduceSum< RAJA_REDUCE_TYPE, realtype> gpu_result(0.0);
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      gpu_result += (xdata[i] * wdata[i] * xdata[i] * wdata[i]);
    }
  );

  /* Reduce across MPI processes */
  realtype sum = static_cast<realtype>(gpu_result);
  SUNMPI_Comm comm = getMPIComm(X);
  return std::sqrt(SUNMPI_Allreduce_scalar(sum, 1, comm));
}

realtype N_VL1Norm_Raja(N_Vector X)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const sunindextype N = getLocalLength(X);

  RAJA::ReduceSum< RAJA_REDUCE_TYPE, realtype> gpu_result(0.0);
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      gpu_result += (abs(xdata[i]));
    }
  );

  /* Reduce across MPI processes */
  realtype sum = static_cast<realtype>(gpu_result);
  SUNMPI_Comm comm = getMPIComm(X);
  return SUNMPI_Allreduce_scalar(sum, 1, comm);
}

void N_VCompare_Raja(realtype c, N_Vector X, N_Vector Z)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(X);
  const sunindextype N = getLocalLength(X);
  realtype *zdata = N_VGetDeviceArrayPointer_Raja(Z);

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      zdata[i] = abs(xdata[i]) >= c ? ONE : ZERO;
    }
  );
}

booleantype N_VInvTest_Raja(N_Vector x, N_Vector z)
{
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(x);
  const sunindextype N = getLocalLength(x);
  realtype *zdata = N_VGetDeviceArrayPointer_Raja(z);

  RAJA::ReduceSum< RAJA_REDUCE_TYPE, realtype> gpu_result(ZERO);
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      if (xdata[i] == ZERO) {
        gpu_result += ONE;
      } else {
        zdata[i] = ONE/xdata[i];
      }
    }
  );

  /* Reduce across MPI processes */
  realtype minimum = static_cast<realtype>(gpu_result);
  SUNMPI_Comm comm = getMPIComm(x);
  realtype global_minimum = SUNMPI_Allreduce_scalar(minimum, 3, comm);

  return (global_minimum < HALF);
}

booleantype N_VConstrMask_Raja(N_Vector c, N_Vector x, N_Vector m)
{
  const realtype *cdata = N_VGetDeviceArrayPointer_Raja(c);
  const realtype *xdata = N_VGetDeviceArrayPointer_Raja(x);
  const sunindextype N = getLocalLength(x);
  realtype *mdata = N_VGetDeviceArrayPointer_Raja(m);

  RAJA::ReduceSum< RAJA_REDUCE_TYPE, realtype> gpu_result(ZERO);
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      bool test = (abs(cdata[i]) > ONEPT5 && cdata[i]*xdata[i] <= ZERO) ||
                  (abs(cdata[i]) > HALF   && cdata[i]*xdata[i] <  ZERO);
      mdata[i] = test ? ONE : ZERO;
      gpu_result += mdata[i];
    }
  );

  /* Reduce across MPI processes */
  realtype sum = static_cast<realtype>(gpu_result);
  SUNMPI_Comm comm = getMPIComm(x);
  realtype global_sum = SUNMPI_Allreduce_scalar(sum, 1, comm);

  return (global_sum < HALF);
}

realtype N_VMinQuotient_Raja(N_Vector num, N_Vector denom)
{
  const realtype *ndata = N_VGetDeviceArrayPointer_Raja(num);
  const realtype *ddata = N_VGetDeviceArrayPointer_Raja(denom);
  const sunindextype N = getLocalLength(num);

  RAJA::ReduceMin< RAJA_REDUCE_TYPE, realtype> gpu_result(std::numeric_limits<realtype>::max());
  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      if (ddata[i] != ZERO)
        gpu_result.min(ndata[i]/ddata[i]);
    }
  );

  /* Reduce across MPI processes */
  realtype minimum = static_cast<realtype>(gpu_result);
  SUNMPI_Comm comm = getMPIComm(num);
  return SUNMPI_Allreduce_scalar(minimum, 3, comm);
}


/*
 * -----------------------------------------------------------------------------
 * fused vector operations
 * -----------------------------------------------------------------------------
 */

int N_VLinearCombination_Raja(int nvec, realtype* c, N_Vector* X, N_Vector z)
{
  hipError_t  err;

  sunindextype N = getLocalLength(z);
  realtype* d_zd = N_VGetDeviceArrayPointer_Raja(z);

  // Copy c array to device
  realtype* d_c;
  err = hipMalloc((void**) &d_c, nvec*sizeof(realtype));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_c, c, nvec*sizeof(realtype), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  // Create array of device pointers on host
  realtype** h_Xd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Xd[j] = N_VGetDeviceArrayPointer_Raja(X[j]);

  // Copy array of device pointers to device from host
  realtype** d_Xd;
  err = hipMalloc((void**) &d_Xd, nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Xd, h_Xd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      d_zd[i] = d_c[0] * d_Xd[0][i];
      for (int j=1; j<nvec; j++)
        d_zd[i] += d_c[j] * d_Xd[j][i];
    }
  );

  // Free host array
  delete[] h_Xd;

  // Free device arrays
  err = hipFree(d_c);
  if (err != hipSuccess) return hipGetLastError();
  err = hipFree(d_Xd);
  if (err != hipSuccess) return hipGetLastError();

  return(0);
}


int N_VScaleAddMulti_Raja(int nvec, realtype* c, N_Vector x, N_Vector* Y, N_Vector* Z)
{
  hipError_t err;

  sunindextype N = getLocalLength(x);
  realtype* d_xd = N_VGetDeviceArrayPointer_Raja(x);

  // Copy c array to device
  realtype* d_c;
  err = hipMalloc((void**) &d_c, nvec*sizeof(realtype));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_c, c, nvec*sizeof(realtype), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  // Create array of device pointers on host
  realtype** h_Yd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Yd[j] = N_VGetDeviceArrayPointer_Raja(Y[j]);

  realtype** h_Zd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Zd[j] = N_VGetDeviceArrayPointer_Raja(Z[j]);

  // Copy array of device pointers to device from host
  realtype** d_Yd;
  err = hipMalloc((void**) &d_Yd, nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Yd, h_Yd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  realtype** d_Zd;
  err = hipMalloc((void**) &d_Zd, nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Zd, h_Zd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      for (int j=0; j<nvec; j++)
        d_Zd[j][i] = d_c[j] * d_xd[i] + d_Yd[j][i];
    }
  );

  // Free host array
  delete[] h_Yd;
  delete[] h_Zd;

  // Free device arrays
  err = hipFree(d_c);
  if (err != hipSuccess) return hipGetLastError();
  err = hipFree(d_Yd);
  if (err != hipSuccess) return hipGetLastError();
  err = hipFree(d_Zd);
  if (err != hipSuccess) return hipGetLastError();

  return(0);
}


/*
 * -----------------------------------------------------------------------------
 * vector array operations
 * -----------------------------------------------------------------------------
 */

int N_VLinearSumVectorArray_Raja(int nvec,
                                 realtype a, N_Vector* X,
                                 realtype b, N_Vector* Y,
                                 N_Vector* Z)
{
  hipError_t err;

  sunindextype N = getLocalLength(Z[0]);

  // Create array of device pointers on host
  realtype** h_Xd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Xd[j] = N_VGetDeviceArrayPointer_Raja(X[j]);

  realtype** h_Yd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Yd[j] = N_VGetDeviceArrayPointer_Raja(Y[j]);

  realtype** h_Zd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Zd[j] = N_VGetDeviceArrayPointer_Raja(Z[j]);

  // Copy array of device pointers to device from host
  realtype** d_Xd;
  err = hipMalloc((void**) &d_Xd, nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Xd, h_Xd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  realtype** d_Yd;
  err = hipMalloc((void**) &d_Yd, nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Yd, h_Yd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  realtype** d_Zd;
  err = hipMalloc((void**) &d_Zd, nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Zd, h_Zd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      for (int j=0; j<nvec; j++)
        d_Zd[j][i] = a * d_Xd[j][i] + b * d_Yd[j][i];
    }
  );

  // Free host array
  delete[] h_Xd;
  delete[] h_Yd;
  delete[] h_Zd;

  // Free device arrays
  err = hipFree(d_Xd);
  if (err != hipSuccess) return hipGetLastError();
  err = hipFree(d_Yd);
  if (err != hipSuccess) return hipGetLastError();
  err = hipFree(d_Zd);
  if (err != hipSuccess) return hipGetLastError();

  return(0);
}


int N_VScaleVectorArray_Raja(int nvec, realtype* c, N_Vector* X, N_Vector* Z)
{
  hipError_t err;

  sunindextype N = getLocalLength(Z[0]);

  // Copy c array to device
  realtype* d_c;
  err = hipMalloc((void**) &d_c, nvec*sizeof(realtype));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_c, c, nvec*sizeof(realtype), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  // Create array of device pointers on host
  realtype** h_Xd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Xd[j] = N_VGetDeviceArrayPointer_Raja(X[j]);

  realtype** h_Zd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Zd[j] = N_VGetDeviceArrayPointer_Raja(Z[j]);

  // Copy array of device pointers to device from host
  realtype** d_Xd;
  err = hipMalloc((void**) &d_Xd, nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Xd, h_Xd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  realtype** d_Zd;
  err = hipMalloc((void**) &d_Zd, nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Zd, h_Zd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      for (int j=0; j<nvec; j++)
        d_Zd[j][i] = d_c[j] * d_Xd[j][i];
    }
  );

  // Free host array
  delete[] h_Xd;
  delete[] h_Zd;

  // Free device arrays
  err = hipFree(d_Xd);
  if (err != hipSuccess) return hipGetLastError();
  err = hipFree(d_Zd);
  if (err != hipSuccess) return hipGetLastError();

  return(0);
}


int N_VConstVectorArray_Raja(int nvec, realtype c, N_Vector* Z)
{
  hipError_t err;

  sunindextype N = getLocalLength(Z[0]);

  // Create array of device pointers on host
  realtype** h_Zd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Zd[j] = N_VGetDeviceArrayPointer_Raja(Z[j]);

  // Copy array of device pointers to device from host
  realtype** d_Zd;
  err = hipMalloc((void**) &d_Zd, nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Zd, h_Zd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      for (int j=0; j<nvec; j++)
        d_Zd[j][i] = c;
    }
  );

  // Free host array
  delete[] h_Zd;

  // Free device arrays
  err = hipFree(d_Zd);
  if (err != hipSuccess) return hipGetLastError();

  return(0);
}


int N_VScaleAddMultiVectorArray_Raja(int nvec, int nsum, realtype* c,
                                     N_Vector* X, N_Vector** Y, N_Vector** Z)
{
  hipError_t err;

  sunindextype N = getLocalLength(X[0]);

  // Copy c array to device
  realtype* d_c;
  err = hipMalloc((void**) &d_c, nsum*sizeof(realtype));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_c, c, nsum*sizeof(realtype), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  // Create array of device pointers on host
  realtype** h_Xd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Xd[j] = N_VGetDeviceArrayPointer_Raja(X[j]);

  realtype** h_Yd = new realtype*[nsum*nvec];
  for (int j=0; j<nvec; j++)
    for (int k=0; k<nsum; k++)
      h_Yd[j*nsum+k] = N_VGetDeviceArrayPointer_Raja(Y[k][j]);

  realtype** h_Zd = new realtype*[nsum*nvec];
  for (int j=0; j<nvec; j++)
    for (int k=0; k<nsum; k++)
      h_Zd[j*nsum+k] = N_VGetDeviceArrayPointer_Raja(Z[k][j]);

  // Copy array of device pointers to device from host
  realtype** d_Xd;
  err = hipMalloc((void**) &d_Xd, nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Xd, h_Xd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  realtype** d_Yd;
  err = hipMalloc((void**) &d_Yd, nsum*nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Yd, h_Yd, nsum*nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  realtype** d_Zd;
  err = hipMalloc((void**) &d_Zd, nsum*nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Zd, h_Zd, nsum*nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      for (int j=0; j<nvec; j++)
        for (int k=0; k<nsum; k++)
          d_Zd[j*nsum+k][i] = d_c[k] * d_Xd[j][i] + d_Yd[j*nsum+k][i];
    }
  );

  // Free host array
  delete[] h_Xd;
  delete[] h_Yd;
  delete[] h_Zd;

  // Free device arrays
  err = hipFree(d_Xd);
  if (err != hipSuccess) return hipGetLastError();
  err = hipFree(d_Yd);
  if (err != hipSuccess) return hipGetLastError();
  err = hipFree(d_Zd);
  if (err != hipSuccess) return hipGetLastError();

  return(0);
}


int N_VLinearCombinationVectorArray_Raja(int nvec, int nsum, realtype* c,
                                         N_Vector** X, N_Vector* Z)
{
  hipError_t err;

  sunindextype N = getLocalLength(Z[0]);

  // Copy c array to device
  realtype* d_c;
  err = hipMalloc((void**) &d_c, nsum*sizeof(realtype));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_c, c, nsum*sizeof(realtype), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  // Create array of device pointers on host
  realtype** h_Xd = new realtype*[nsum*nvec];
  for (int j=0; j<nvec; j++)
    for (int k=0; k<nsum; k++)
      h_Xd[j*nsum+k] = N_VGetDeviceArrayPointer_Raja(X[k][j]);

  realtype** h_Zd = new realtype*[nvec];
  for (int j=0; j<nvec; j++)
    h_Zd[j] = N_VGetDeviceArrayPointer_Raja(Z[j]);

  // Copy array of device pointers to device from host
  realtype** d_Xd;
  err = hipMalloc((void**) &d_Xd, nsum*nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Xd, h_Xd, nsum*nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  realtype** d_Zd;
  err = hipMalloc((void**) &d_Zd, nvec*sizeof(realtype*));
  if (err != hipSuccess) return hipGetLastError();
  err = hipMemcpy(d_Zd, h_Zd, nvec*sizeof(realtype*), hipMemcpyHostToDevice);
  if (err != hipSuccess) return hipGetLastError();

  RAJA::forall< RAJA_NODE_TYPE >(RAJA::RangeSegment(zeroIdx, N),
    RAJA_LAMBDA(sunindextype i) {
      for (int j=0; j<nvec; j++) {
        d_Zd[j][i] = d_c[0] * d_Xd[j*nsum][i];
        for (int k=1; k<nsum; k++) {
          d_Zd[j][i] += d_c[k] * d_Xd[j*nsum+k][i];
        }
      }
    }
  );

  // Free host array
  delete[] h_Xd;
  delete[] h_Zd;

  // Free device arrays
  err = hipFree(d_Xd);
  if (err != hipSuccess) return hipGetLastError();
  err = hipFree(d_Zd);
  if (err != hipSuccess) return hipGetLastError();

  return(0);
}


/*
 * -----------------------------------------------------------------
 * Enable / Disable fused and vector array operations
 * -----------------------------------------------------------------
 */

int N_VEnableFusedOps_Raja(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  if (tf) {
    /* enable all fused vector operations */
    v->ops->nvlinearcombination = N_VLinearCombination_Raja;
    v->ops->nvscaleaddmulti     = N_VScaleAddMulti_Raja;
    v->ops->nvdotprodmulti      = NULL;
    /* enable all vector array operations */
    v->ops->nvlinearsumvectorarray         = N_VLinearSumVectorArray_Raja;
    v->ops->nvscalevectorarray             = N_VScaleVectorArray_Raja;
    v->ops->nvconstvectorarray             = N_VConstVectorArray_Raja;
    v->ops->nvwrmsnormvectorarray          = NULL;
    v->ops->nvwrmsnormmaskvectorarray      = NULL;
    v->ops->nvscaleaddmultivectorarray     = N_VScaleAddMultiVectorArray_Raja;
    v->ops->nvlinearcombinationvectorarray = N_VLinearCombinationVectorArray_Raja;
  } else {
    /* disable all fused vector operations */
    v->ops->nvlinearcombination = NULL;
    v->ops->nvscaleaddmulti     = NULL;
    v->ops->nvdotprodmulti      = NULL;
    /* disable all vector array operations */
    v->ops->nvlinearsumvectorarray         = NULL;
    v->ops->nvscalevectorarray             = NULL;
    v->ops->nvconstvectorarray             = NULL;
    v->ops->nvwrmsnormvectorarray          = NULL;
    v->ops->nvwrmsnormmaskvectorarray      = NULL;
    v->ops->nvscaleaddmultivectorarray     = NULL;
    v->ops->nvlinearcombinationvectorarray = NULL;
  }

  /* return success */
  return(0);
}


int N_VEnableLinearCombination_Raja(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvlinearcombination = N_VLinearCombination_Raja;
  else
    v->ops->nvlinearcombination = NULL;

  /* return success */
  return(0);
}

int N_VEnableScaleAddMulti_Raja(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvscaleaddmulti = N_VScaleAddMulti_Raja;
  else
    v->ops->nvscaleaddmulti = NULL;

  /* return success */
  return(0);
}

int N_VEnableLinearSumVectorArray_Raja(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvlinearsumvectorarray = N_VLinearSumVectorArray_Raja;
  else
    v->ops->nvlinearsumvectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableScaleVectorArray_Raja(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvscalevectorarray = N_VScaleVectorArray_Raja;
  else
    v->ops->nvscalevectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableConstVectorArray_Raja(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvconstvectorarray = N_VConstVectorArray_Raja;
  else
    v->ops->nvconstvectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableScaleAddMultiVectorArray_Raja(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvscaleaddmultivectorarray = N_VScaleAddMultiVectorArray_Raja;
  else
    v->ops->nvscaleaddmultivectorarray = NULL;

  /* return success */
  return(0);
}

int N_VEnableLinearCombinationVectorArray_Raja(N_Vector v, booleantype tf)
{
  /* check that vector is non-NULL */
  if (v == NULL) return(-1);

  /* check that ops structure is non-NULL */
  if (v->ops == NULL) return(-1);

  /* enable/disable operation */
  if (tf)
    v->ops->nvlinearcombinationvectorarray = N_VLinearCombinationVectorArray_Raja;
  else
    v->ops->nvlinearcombinationvectorarray = NULL;

  /* return success */
  return(0);
}

} // extern "C"
